#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 16

__global__ void fill_random(float *A, int N, size_t pitch) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    //A[j + N * i] = i + j;
    float *row = (float *)((char *) A + i * pitch);
    row[j] = i + j;
}

void mult(float *A, float *B, float *C, size_t N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            C[i * N + j] = 0;
            for (int k = 0; k < N; k++) {
                C[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
}

__global__ void matMult(float *A, size_t pitchA, float *B, size_t pitchB,
                        float *C, size_t pitchC, size_t N) {
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = N * BLOCK_SIZE * by;
    int bBegin = BLOCK_SIZE * bx;
    int aEnd = aBegin + N - 1;

    int aStep = BLOCK_SIZE;
    int bStep = BLOCK_SIZE * N;

    float sum = 0.0f;

    for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep) {
        __shared__ float as[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float bs[BLOCK_SIZE][BLOCK_SIZE];

        float *rowA = (float *)((char *) A + pitchA * ty);
        float *rowB = (float *)((char *) B + pitchB * ty);
        as[ty][tx] = rowA[ia + tx];
        bs[ty][tx] = rowB[ib + tx];

        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            sum += as[ty][k] * bs[k][tx];
        }
        __syncthreads();
    }

    int ic = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    float *rowC = (float *)((char *) C + pitchC * ty);
    rowC[ic + tx] = sum;
}

int main(int argc, char **argv) {
    int N = BLOCK_SIZE;
    if (argc > 1) {
        N = atoi(argv[1]);
    }

    std::clog << "Given matrix size: " << N << std::endl;

    /// Round up to the next power of 2 (min is 16).
    N = N < BLOCK_SIZE ? BLOCK_SIZE : pow(2, ceil(log2(N)));
    
    std::clog <<  "Current matrix size: " << N << std::endl;

    hipSetDevice(0);

    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *A = new float[N * N];
    float *C = new float[N * N];
    float *C2 = new float[N * N];
    float *f = new float[N];
    float *x = new float[N];
    for (int i = 0; i < N; i++) {
        f[i] = i;
        x[i] = 0;
    }

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = 0;
        }
    }
    float *ADev,  *CDev, *fDev, *xDev;
    size_t pitchA, pitchC;
    hipMallocPitch((void **) &ADev, &pitchA, N * sizeof(float), N);
    hipMallocPitch((void **) &CDev, &pitchC, N * sizeof(float), N);
    hipMalloc((void **) &fDev, N * sizeof(float));
    hipMalloc((void **) &xDev, N * sizeof(float));
    hipMemcpy2D(ADev, pitchA, A, N * sizeof(float), N * sizeof(float), N, hipMemcpyHostToDevice);
    hipMemcpy(fDev, f, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(xDev, x, N * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    fill_random<<<numBlocks, threadsPerBlock>>>(ADev, N, pitchA);
    matMult<<<numBlocks, threadsPerBlock>>>(ADev, pitchA, ADev, pitchA, CDev, pitchC, N);

    hipMemcpy2D(A, N * sizeof(float), ADev, pitchA, N * sizeof(float), N, hipMemcpyDeviceToHost);
    hipMemcpy2D(C, N * sizeof(float), CDev, pitchC, N * sizeof(float), N, hipMemcpyDeviceToHost);
    hipMemcpy(f, fDev, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(x, xDev, N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
/*
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + i] += A[i * N + j];
        }
    }*/
    hipFree(ADev);
    hipFree(CDev);
    hipFree(fDev);
    hipFree(xDev);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    mult(A, A, C2, N);

    printf("Elapsed time: %.2f ms\n", gpuTime);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%3.0f ", A[i * N + j]);
        }
        printf("| %3.0f\n", f[i]);
    }
    printf("multiply:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%3.0f ", C[i * N + j]);
        }
        printf("\n");
    }
    printf("right:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%3.0f ", C2[i * N + j]);
        }
        printf("\n");
    }
    delete A;
    delete f;
    delete x;
    return 0;
}